#include "hip/hip_runtime.h"
#include"Header_Enface.h"

__global__ void avgKernel(unsigned short* src_Buffer, float* dst_Buffer, int frame_num, int num_Frames, int frameSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ unsigned short tempVal;
	tempVal = 0;
	for (int i = 0; i < num_Frames; i++)
	{
		tempVal += src_Buffer[(frame_num + i) * frameSize + idx];
	}

	dst_Buffer[idx] = (float)tempVal / num_Frames;
	 
}



 
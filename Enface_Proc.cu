#include"Header_Enface.h"
void frameAvg(unsigned short* dev_multiFrameBuff,float* dev_displayBuff, int width, int height, int numberOfFrames, int frameNum)
{
	int numThreadsPerBlock = 256;

	dim3 dimBlockX(numThreadsPerBlock);
	dim3 dimGridX((width * height)/dimBlockX.x);
	avgKernel << <dimGridX, dimBlockX >> > (dev_multiFrameBuff, dev_displayBuff, frameNum, numberOfFrames, width * height);
	
}	

 


void writeMatToFile(cv::Mat m, const char* filename)
{
	ofstream fout(filename);

	if (!fout)
	{
		cout << "File Not Opened" << endl;  return;
	}

	for (int i = 0; i < m.rows; i++)
	{
		for (int j = 0; j < m.cols; j++)
		{
			fout << m.at<float>(i, j) << "\t";
		}
		fout << endl;
	}

	fout.close();
}

void enface(float* d_A, float* d_x, float*d_y, const int row, const int col)
{
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	float alf = 0.00102;
	float beta = 0;
	stat = hipblasCreate(&handle);
	stat = hipblasSgemv(handle, HIPBLAS_OP_T, col, row, &alf, d_A, col, d_x, 1, &beta, d_y, 1);//swap col and row
	

	


	hipblasDestroy(handle);
}